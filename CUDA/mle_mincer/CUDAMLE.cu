#include "hip/hip_runtime.h"
//  Created by Rodrigo Azuero on 2/24/16.
//  Copyright (c) 2016 Rodrigo Azuero Melo. All rights reserved.
//


#include <algorithm>
#include <ctime>
#include <cmath>
#include <fstream>
#include <iostream>
#include <iomanip>
#include <math.h>
#include <sstream>
#include <string>
#include <stdlib.h>
#include <stdio.h>
#include <vector>
#include <unistd.h>
#include <stdio.h>
#include <errno.h>
#include <gsl/gsl_sf_bessel.h>
#include <gsl/gsl_roots.h>
#include <gsl/gsl_matrix.h>
#include <gsl/gsl_blas.h>
#include <gsl/gsl_math.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/random/linear_congruential_engine.h>
#include <thrust/random/normal_distribution.h>
#include <unistd.h>
#include <nlopt.hpp>
using std::vector;
using namespace std;


__device__ __host__ float normpdfESTANDAR(float D){
        float term=D*D;
        float exp=-term;
        float num=expf(exp);
        float den=powf(2*3.141592,0.5);
        float ans=num/den;
        return(ans);
}
//----------
//0.0. Wages
//------------
//===================================================================================//


__device__ __host__  float F_predwageD(float bbeta0, float bbeta1, float bbeta2, float bbeta3,  float Schooling, float Age){
    //1. Defining the output
    float wage=bbeta0+bbeta1*Schooling+bbeta2*Age+bbeta3*Age*Age;
    return(wage);
}



//=================================================================================
//1.0. Likelihood of Wages
//=================================================================================
__host__ __device__ float F_likelihood_wageD(float bbeta0, float bbeta1, float bbeta2, float bbeta3, float stdwage, float Schooling, float Age, float Wage){
    
    //1. Obtaining the predicted Wage:
    float predwage=F_predwageD(bbeta0,bbeta1,bbeta2,bbeta3,Schooling,Age);
    float likelihood_wage=Wage-predwage;
    likelihood_wage=likelihood_wage/stdwage;
    likelihood_wage=normpdfESTANDAR(likelihood_wage);
    
    likelihood_wage=likelihood_wage/stdwage;
    if(likelihood_wage==0){
        likelihood_wage=0.000001;
    }
    likelihood_wage=logf(likelihood_wage);
    return(likelihood_wage);
}



//----------------------------------------------------
//2. Block of defining likelihood for all the dataset
//---------------------------------------------------
 
__device__ __host__ float F_likelihoodSCALARD(const float* par_likelihood,
                                                float  Schooling,
                                                float Age,
                                                float Wage){


     //float aalpha1m=FT_expD(par_likelihood[0]);
    float bbeta0=(par_likelihood[0]);
    float bbeta1=(par_likelihood[1]);
    float bbeta2=(par_likelihood[2]);
    float bbeta3=(par_likelihood[3]);
    float std=exp(par_likelihood[4]);

    float iilike=F_likelihood_wageD(bbeta0, bbeta1,  bbeta2,  bbeta3,  std,  Schooling,  Age,  Wage);
    return(iilike);
}
    

//Functions of loading all the likelihoods. Looping over individuals for the function F_likelihoodSCALARD
__global__ void KernelLikelihood(
    const float* par_likelihood, 
    const float *V_Schooling,
    const float *V_Age,
    const float * V_Wage, float *output){

    //Indexing
    int ii=threadIdx.x+blockIdx.x*blockDim.x;
    if(ii<1000){
        //Loading
        float Schooling=V_Schooling[ii];
        float Age=V_Age[ii];
        float Wage=V_Wage[ii];
        float iilik=F_likelihoodSCALARD(par_likelihood,Schooling,Age,Wage);
        output[ii]=iilik;
    }
    if(ii>=1000){
        output[ii]=0;
    }
}






//Attempt of likelihood
float F_likelihood_FIN(const double *PARDOUBLE){ //In this function I will call the kernel
    //0. Define the current directory


    int THREADSPERBLOCK=200;
    int TOTALTHREADS=800;
    int SIZEOBS=800;
    int NVAR=3;
    int NPAR=5;
    //Passing copies of the parameters to the device
    float *Schooling, *Age, *Wage, *output, *PAR;
    float *d_Schooling, *d_Age, *d_Wage, *d_output, *d_PAR;
    
    size_t sizePARFLOAT=NPAR*sizeof(float);
    size_t dimobsFLOAT=TOTALTHREADS*sizeof(float);


    //Allocate space for device copies of Parameters
    hipMalloc((void **)&d_PAR,sizePARFLOAT);
    hipMalloc((void **)&d_Schooling,dimobsFLOAT);
    hipMalloc((void **)&d_Age,dimobsFLOAT);
    hipMalloc((void **)&d_Wage,dimobsFLOAT);
    hipMalloc((void **)&d_output,dimobsFLOAT);
    
    //*Cliveswithfather12, *Cliveswithmother12, *Hhchores12, *Meffort12, *Feffort12, *Mage12, *Fage12, *Cedad_meses12, *Ctestsfactorsss2012, *Ctestsfactor2ss_10, *Myrschool12, *Fyrschool12, *Ffraclabor12, *Mfraclabor12, *Mwage12, *Fwage12, *Mnly12, *Fnly12, *Cfactorinv12, *Cchildcare12, *Ccareskills12, *Hbarg, *Feffort10, *Meffort10, *Cfactorinv10, *Cbirthfactor, *Mwage10, *Fwage10, *Mnly10, *Fnly10, *Cedad_meses10, *Cliveswithfather10, *Cliveswithmother10, *Mage10, *Fage10, *Mfraclabor10, *Ffraclabor10, *Cchildcare10, *Hchildcareobs, *PG, *MTJH,*MRATIO,*Unemployment, *Wageratio,*Distance,*Magegroup10,*Magegroup12,*Hmemberstotal10,*Hmemberstotal12,*CSTDtepsi_pb_coo10,*CSTDtepsi_pb_len10,*CSTDtepsi_pb_mot10,*CSTDtvip_pb10,*CSTDcbcl1_pb_110,*CSTDcbcl1_pb_210,*CSTDcbcl1_pb_310,*CSTDcbcl1_pb_410,*CSTDcbcl1_pb_510,*CSTDcbcl1_pb_610,*CSTDcbcl1_pb_710,*CSTDtadi_pb_cog12, *CSTDtadi_pb_mot12, *CSTDtadi_pb_len12, *CSTDtadi_pb_se12, *CSTDbt_112, *CSTDbt_212, *CSTDbt_312, *CSTDbt_412, *CSTDbt_512, *CSTDbt_t12,  *CSTDhtks_st12, *CSTDbdst_st12, *CSTDppvt_t12,  *Ccondpregg7b, *Ccondpregg8b, *Ccondpregg9, *Ccondpregg11b, *Ccondpregg24, *Ccondpregg23,*Cwais_pb_num, *Cwais_pb_vo, *Cbfi_pb_ama,  *Cbfi_pb_ape, *Cbfi_pb_ext, *Cbfi_pb_neu, *Cbfi_pb_res, *Cpsi_pb_total,*Hbargg2a, *Hbargg2b, *Hbargg2c, *Hbargg2d, *Hbargg2e, *Hbargg2f, *Hbargg2g, *Hbargg2h, *Hbargg2i, *Hbargg2j, *Hcaresacwom, *Hcaresacman, *Cinvf11a, *Cinvf11b,  *Cinvf11c, *Cinvf11d, *Cinvf11e, *Cinvf11f, *Cinvf11g, *Cinvf11h, *Cinvf11i, *Cinvf11j, *Cinvf11k,*Csharesbedroomhowmany12,  *Csharesbedhowmany12, *g42_a2, *g42_b2, *g42_c2, *g42_d2, *g42_e2, *g42_f2,*g42_a1, *g42_b1, *g42_c1, *g42_d1, *g42_e1, *g42_f1,*f21a_p_t, *f21b_p_t, *f21c_p_t, *f21d_p_t, *f21e_p_t, *f21f_p_t, *f21g_p_t, *f21h_p_t, *f21i_p_t, *f21j_p_t, *f21k_p_t, *f21l_p_t, *f21m_p_t, *f21n_p_t,*f21a_m_t, *f21b_m_t, *f21c_m_t, *f21d_m_t, *f21e_m_t, *f21f_m_t, *f21g_m_t, *f21h_m_t, *f21i_m_t, *f21j_m_t, *f21k_m_t, *f21l_m_t, *f21m_m_t, *f21n_m_t;
    
    
    Schooling=(float *)malloc(dimobsFLOAT);
    Age=(float *)malloc(dimobsFLOAT);
    Wage=(float *)malloc(dimobsFLOAT);
    output=(float *)malloc(dimobsFLOAT); 
    PAR=(float *)malloc(sizePARFLOAT);
    


    //Loading the dataset
    std::ifstream theFile ("DATACUDA.csv");
    double MYARRAY[SIZEOBS+1][NVAR+1];

    std::string line;
    std::vector<std::vector<std::string> > values;
    int it=0;
    int it2=0;
    std::string line_value;
    std::vector<std::string> line_values;
    std::stringstream ss;
    while(std::getline(theFile, line))
    {
        
        ss<<line;
        //std::stringstream ss(line);
        //std::string item;
        //cout <<  << "linevalprev"<<endl;
        while(std::getline(ss, line_value, ','))
        {
            line_values.push_back(line_value);
            MYARRAY[it][it2] = ::atof(line_value.c_str());
            
            //MYARRAY[it][it2]=std::stod (line_value); only for c++11 compi
            it2=it2+1;
            if (it2==NVAR){ //later change 4 for
                it2=0;
            }
        }
        values.push_back(line_values);
        
        //For c++11 used values.emplace_back(line_values);
        
        //cout << line_value<< "line_value2"<< endl;
        it=it+1;
        //Free the string types
        line_value.clear();
        line_values.clear();
        ss.clear();
        
    }

    //Allocate initialization 
    for (int it=0; it<SIZEOBS;it++){
        Age[it]=MYARRAY[it][0];
        Schooling[it]=MYARRAY[it][1];
        Wage[it]=MYARRAY[it][2];
        
    }//Finishmloading data
    cout << " ---- " << endl;
    //Allocate parameters
    for (int it=0; it<NPAR;it++){
        PAR[it]=(float)(PARDOUBLE[it]);
        cout <<PAR[it] << " par[it]"<< endl;
        cout << it  << " it "<< endl;
    }
    cout << " ---- " << endl;
    //Cuda copies double
    hipMemcpy(d_Schooling,Schooling,dimobsFLOAT,hipMemcpyHostToDevice);
    hipMemcpy(d_Age,Age,dimobsFLOAT,hipMemcpyHostToDevice); 
    hipMemcpy(d_Wage,Wage,dimobsFLOAT,hipMemcpyHostToDevice); 
    hipMemcpy(d_PAR,PAR,sizePARFLOAT,hipMemcpyHostToDevice);

    //Run kernel
    KernelLikelihood<<<TOTALTHREADS/THREADSPERBLOCK,THREADSPERBLOCK>>>(d_PAR,d_Schooling,d_Age,d_Wage,d_output);

    hipMemcpy(output,d_output,dimobsFLOAT,hipMemcpyDeviceToHost);
    free(Wage);
    free(output);
    free(Age);
    free(Schooling);
    free(PAR);
    hipFree(d_Wage);
    hipFree(d_output);
    hipFree(Age);
    hipFree(d_Schooling);
    hipFree(d_PAR);
    float sumA=0;
    for(int ii=0; ii<SIZEOBS;ii=ii+1){
        sumA+=output[ii];
        //cout << output[ii] << " output[ii] " << endl;

        //cout << Fwagepred[ii] << "wagepredii"<<endl;
    }
    cout << sumA << " FLIK" << endl;
    sumA=-sumA;
    return(sumA);
}


int iterat=0;
double FLIKMINIMIZED(unsigned n, const double *x, double *grad, void *FLIKMNIMZED_data){
    double result=F_likelihood_FIN(x);
    cout << result << " evalF"<< endl;
    iterat=iterat+1;
    printf("Iteration=(%d); Feval=%0.10g\n", iterat, result);
    return(result);
}


int  main (const int           argc,
           const char * const  argv[])

{
	

        std::string line;
        std::vector<std::vector<std::string> > values;
        vector<double> PARRA;
        PARRA.resize(5);
        std::ifstream PARAMETERS ("INITIALGUESS.csv");
        
        std::string linePARAM;
        int itpar=0;
        while(std::getline(PARAMETERS, line))
        {
            PARRA[itpar]=::atof (line.c_str());
            //PAR[itpar]=std::stod (line); only C++11s
            
            
            
            itpar=itpar+1;
        }
        //And the ones that were not stored
        
        //Loading The parameters into PARDOUBLE
        double PARDOUBLE[5]={};
        for (int ii=0;ii<5;ii=ii+1){
            PARDOUBLE[ii]=PARRA[ii];
        }
        
        
        cout << " before evaluating likelihood  " << endl;
        
        cout << F_likelihood_FIN(PARDOUBLE) << " LIKELIHOOD " << endl;
        int optimize=1;
        if(optimize==1){
            nlopt_opt opt3;
            opt3 = nlopt_create(NLOPT_LN_SBPLX, 5); /* algorithm and dimensionality */
            nlopt_set_min_objective(opt3, FLIKMINIMIZED, NULL);
            //nlopt_set_xtol_rel(opt3, 1.0e-4);
            //nlopt_set_maxtime(opt3,  10000);
            nlopt_set_maxeval(opt3,2000);
            double x3[5]={0};
            for(int it=0;it<5;it=it+1){
                x3[it]=PARDOUBLE[it];
            }
            
            double minf3; /* the minimum objective value, upon return */
            nlopt_optimize(opt3, x3, &minf3);
            if (nlopt_optimize(opt3, x3, &minf3) < 0) {
                printf("nlopt failed!\n");
            }
            else {
                printf("found minimum at f(%g) = %0.10g\n", x3[0], minf3);
            }
            
            //Saving the vector of optimal parameters in csv file
            ofstream optparam("PARAMETERSFOUND.csv");
            for (int it=0;it<5;it=it+1){
                optparam<<x3[it] << endl;
            }
            optparam.close();
            
            
            nlopt_destroy(opt3);
        }
    

    



	return 0;
    
}